#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <time.h>

#define R_MIN 1.0
#define R_MAX -1.0

#define in_s 28
#define k_s 5
#define out_s 24

void convolve(double * input, double * output, double * kernel, double bias, int stride, int in_size, int out_size, int kernel_size);
void init_mat(double * input, int len);    // Generate 2-D matrix, give allocated pointer
double rand_double(double min, double max);     // Generate random float in interval
void compare_mat(double * A, double * B, int r_len, int c_len); // Matrix comparison

/* --- Interval code --- */
double interval(struct timespec start, struct timespec end)
{
  struct timespec temp;
  temp.tv_sec = end.tv_sec - start.tv_sec;
  temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  if (temp.tv_nsec < 0) {
    temp.tv_sec = temp.tv_sec - 1;
    temp.tv_nsec = temp.tv_nsec + 1000000000;
  }
  return (((double)temp.tv_sec) + ((double)temp.tv_nsec)*1.0e-9);
}

/* --- CUDA functions --- */


/* --- End CUDA functions --- */

int main(int argc, char **argv) {
    // Init vars
    double * in_h, * out_h, * ker_h;
    double * in_d, * out_d, * ker_d;
    double * out_CPU;	// CPU output matrix
    struct timespec time_start, time_stop;
    // struct timespec time_start_2, time_stop_2;

    // Allocate arrays
    int in_alloc_size = sizeof(double) * in_s * in_s;
    int out_alloc_size = sizeof(double) * out_s * out_s;
    int ker_alloc_size = sizeof(double) * k_s * k_s;

    in_h = (double *) malloc(in_alloc_size);
    out_h = (double *) malloc(out_alloc_size);
    ker_h = (double *) malloc(ker_alloc_size);
	out_CPU = (double *) malloc(out_alloc_size);

    hipMalloc((void **) &in_d, in_alloc_size);
    hipMalloc((void **) &out_d, out_alloc_size);
    hipMalloc((void **) &ker_d, ker_alloc_size);

    // Init matrix
    init_mat(in_h, in_s);
    init_mat(ker_h, k_s);

    // Copy data to GPU
    hipMemcpy(in_d, in_h, in_alloc_size, hipMemcpyHostToDevice);
    hipMemcpy(ker_d, ker_h, ker_alloc_size, hipMemcpyHostToDevice);

    // Run kernel

    // Copy back
    hipMemcpy(out_h, out_d, out_alloc_size, hipMemcpyDeviceToHost);
    printf("Done with Conv on GPU\n");

    // Deallocate GPU mem
    hipFree(in_d);
    hipFree(out_d);
    hipFree(ker_d);

    // Run CPU code

    // Compare accuracy
    compare_mat(out_CPU, out_h, out_s);

    // Compare time


}

void convolve(double * input, double * output, double * kernel, double bias, int stride,  int in_size, int out_size, int kernel_size) {
    int i, j, ii, jj, ki, kj;
    double sum;
    // For each output entry:
    for (i = 0; i < out_size; i++) {
        for (j = 0; j < out_size; j++) {
            sum = 0;
            ki = 0; kj = 0;

            // Convolve
            for (ii = i; ii < i + kernel_size; ii++) {
                kj = 0;
                for (jj = j; jj < j + kernel_size; jj++) {
                    sum += kernel[ki * kernel_size + kj] * input[ii * in_size + jj];
                    kj++;
                }
                ki++;
            }

            output[i * out_size + j] = sum + bias;
        }
    }

}

double rand_double(double min, double max) {
    double random = ((double) rand()) / (double) RAND_MAX;
    double diff = max - min;
    double r = random * diff;
    return min + r;
}

void init_mat(double * input, int len) {
    for (int i = 0; i < len; i++) {
        for (int j = 0; j < len; j++) {
            input[i * len + j] = rand_double(R_MIN, R_MAX);
        }
    }
}

void compare_mat(double * A, double * B, int r_len, int c_len) {
    double avg_err;
    double largest_err = 0;
    double total_err = 0;

    for (int i = 0; i < c_len; i++) {
        for (int j = 0; j < r_len; j++) {
            double err = abs(A[i * r_len + j] - B[i * r_len + j]);
            total_err += err;
            if (err > largest_err) largest_err = err;
        }
    }

    avg_err = total_err / (r_len * c_len);

    if (total_err == 0) printf("No error!\n");
    else {
        printf("Errors founded!\n");
        printf("Average absolute err per element is: %f\n", avg_err);
        printf("Largest absolute err between 2 elements was: %f\n", largest_err);
        printf("Total err is: %f\n", total_err);
    }
}